#include "hip/hip_runtime.h"

#include <sys/time.h>
#include <time.h>
#include <unistd.h>

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cstdio>

#define DEBUG_CUDA 0
#define TESTPERFORMANCE_NO_LEVENSHTEIN 1

#define MIN3(a, b, c) \
    ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

__global__ void searchPattern(d_buf, n_bytes, pattern, nb_patterns, lastPatternAnalyzedByGPU, sizePatterns, numbersOfMatch, indexFinishMyPieceWithoutExtra, myRank, numberProcesses, indexStartMyPiece, approx_factor){

    // I analyze the second half of the patterns
    // for (i = 0; i < lastPatternAnalyzedByGPU; i++) {
    if(i < lastPatternAnalyzedByGPU){
        double timestampStart;
        double timestampFinish;

#if DEBUG
        printf(
                        "----- MPI %d (out of %d) & OpenMP %d (out of %d). Started "
                        "to analize pattern n° %d.\n",
                        myRank, numberProcesses, omp_get_thread_num(),
                        omp_get_num_threads(), i);
#endif

        int size_pattern = strlen(pattern[i]);
        int *column;

        column = (int *) malloc((size_pattern + 1) * sizeof(int));
        if (column == NULL) {
            fprintf(
                    stderr,
                    "Error: unable to allocate memory for column (%ldB)\n",
                    (size_pattern + 1) * sizeof(int));
            // return 1;
        }

        // If I am not the last rank I should take in consideration
        // extra characters from the next piece: in this way I don't
        // miss words which are placed between two pieces. If am the
        // last rank I don't take extra characters as the other ranks
        // since the file is finished.
        int indexFinishMyPieceWithExtra =
                indexFinishMyPieceWithoutExtra;
        if (myRank != numberProcesses - 1) {
            indexFinishMyPieceWithExtra += size_pattern - 1;
        }

#if DEBUG
        printf(
                        "Rank %d. I received the info from rank 0. Start index: "
                        "%d. Finish index: %d\n",
                        myRank, indexStartMyPiece, indexFinishMyPieceWithoutExtra);
                    printf("Rank %d. Final index updated: %d.\n", myRank,
                           indexFinishMyPieceWithExtra);
#endif

#if DEBUGPIECEREAD
        printf("Rank %d: I will read the following text:\n", myRank);
                    int j;
                    for (j = indexStartMyPiece;
                         j < indexFinishMyPieceWithExtra - approx_factor; j++) {
                        printf("%c", buf[j]);
                    }
                    printf("\n");
#endif

        // Traverse the input data up to the end of the file
        n_bytes = indexFinishMyPieceWithExtra;

#if DEBUG
        printf(
                        "----- MPI %d (out of %d) & OpenMP %d (out of %d). Index "
                        "Start: %d. Index finish: %d.\n",
                        myRank, numberProcesses, omp_get_thread_num(),
                        omp_get_num_threads(), indexStartMyPiece, n_bytes);
#endif

        timestampStart = omp_get_wtime();

        // It's not possible to parallelize with OpenMP this for since
        // the cycles are interconnected.
        int r;
        for (r = indexStartMyPiece; r < n_bytes - approx_factor; r++) {
#if DEBUGBYTEOPENMP
            printf(
                            "MPI %d (out of %d) & OpenMP %d (out of %d). I am "
                            "analyzing byte %d for pattern %d\n",
                            myRank, numberProcesses, omp_get_thread_num(),
                            omp_get_num_threads(), r, i);

#endif

#if DEBUGCHARACTERS
            printf("Rank %d. I read the character: %c \n", myRank,
                               buf[j]);
#endif

            int distance = 0;
            int size;
            size = size_pattern;
            if (n_bytes - r < size_pattern) {
                size = n_bytes - r;
            }

#if DEBUGOPENMPPOINTERS
            printf(
                            "Pattern: %p. Buf: %p. Size: %p. Columns: %p.\ni "
                            "address: %p. i value: %d. j address: %p. j value: %d "
                            "\n",
                            &pattern, &buf[j], &size, &column, &i, i, &j, j);
#endif
            // distance = levenshtein(pattern[i], &buf[r], size, column);

            unsigned int x, y, lastdiag, olddiag;

#pragma unroll
            for (y = 1; y <= size; y++) {
                column[y] = y;
            }
#pragma unroll
            for (x = 1; x <= size; x++) {
                column[0] = x;
                lastdiag = x - 1;
                for (y = 1; y <= size; y++) {
                    olddiag = column[y];
                    column[y] = MIN3(
                            column[y] + 1, column[y - 1] + 1,
                            lastdiag + (pattern[y - 1] == (&buf[j])[x - 1] ? 0 : 1));
                    lastdiag = olddiag;
                }
            }

            distance = column[size];

            if (distance <= approx_factor) {
                numbersOfMatch[i] += 1;

#if DEBUG
                printf("Rank %d. MATCH FOUND! \n", myRank);
#endif
            }
        }
        timestampFinish = omp_get_wtime();

#if DEBUG
        double elapsedTime = timestampFinish - timestampStart;
                    printf("Time elapsed for a thread: %g.\n", elapsedTime);
#endif
        free(column);
    }

}


extern "C" int * searchPatternsInPieceDatabase(char *buf, int n_bytes, char **pattern, int nb_patterns, int lastPatternAnalyzedByGPU, int *sizePatterns, int indexFinishMyPieceWithoutExtra, int myRank, int numberProcesses, int indexStartMyPiece, int approx_factor){

    // Allocate arrays in device memory

    int numbersOfMatch[nb_patterns];

    char *d_buf;
    hipMalloc(&d_buf, n_bytes);
    hipMemcpy(d_buf, buf, n_bytes, hipMemcpyHostToDevice);

    int *d_sizePatterns;
    hipMalloc(&d_sizePatterns, nb_patterns * sizeof(int));

    hipMemcpy(d_sizePatterns, sizePatterns, nb_patterns, hipMemcpyHostToDevice);

    int *d_numbersOfMatch;
    hipMalloc(&d_numbersOfMatch, nb_patterns * sizeof(int));

#if DEBUG_CUDA
    printf("DEBUG_CUDA: Starting memory transfers...\n");
#endif

    char *d_pattern;
    hipMalloc(&d_pattern, nb_patterns * sizeof(char *));

    // Allocando spazio per pattern
    for(int i = 0; i < nb_patterns; i++){
        char * patternInUse = &d_pattern[i];
        hipMalloc(&patternInUse, sizePatterns[i] * sizeof(char));
        hipMemcpy(&d_pattern[i], pattern[i], sizePatterns[i], hipMemcpyHostToDevice);
    }

    // Initialize array
    for(int i = 0; i < nb_patterns; i++){
        d_numbersOfMatch = 0;
    }

    int sizeGrid = 256;
    int sizeBlocks = 1;

    searchPattern<<<sizeGrid, sizeBlocks>>>(d_buf, n_bytes, pattern, nb_patterns, lastPatternAnalyzedByGPU, sizePatterns, numbersOfMatch, indexFinishMyPieceWithoutExtra, myRank, numberProcesses, indexStartMyPiece, approx_factor);

    hipMemcpy(numbersOfMatch, d_numbersOfMatch, nb_patterns * sizeof(int),
               hipMemcpyDeviceToHost);

    return numbersOfMatch;

}


#include <sys/time.h>
#include <time.h>
#include <unistd.h>

// CUDA-C includes
#include <hip/hip_runtime.h>


#include <cstdio>

#define DEBUG_CUDA 0
#define TESTPERFORMANCE_NO_LEVENSHTEIN 0

#define MIN3(a, b, c) \
    ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

int *d_numbersOfMatch;

__global__ void
searchPattern(char *buf, int n_bytes, char **pattern, int nb_patterns, int lastPatternAnalyzedByGPU, int *sizePatterns,
              int *numbersOfMatch, int indexFinishMyPieceWithoutExtra, int myRank, int numberProcesses,
              int indexStartMyPiece, int approx_factor) {


    // I analyze the second half of the patterns
    // for (i = 0; i < lastPatternAnalyzedByGPU; i++) {

    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < lastPatternAnalyzedByGPU) {

        if (TESTPERFORMANCE_NO_LEVENSHTEIN) {
            // Sleep 1 microsecond

            // This works just with Compute Capability > 7.0
            // unsigned int ns = 1000;
            // __nanosleep(ns);

            /*

            Without the possibility to use nanosleep the only thing that it's possible to do is to wait an arbitrary number of clocks. But we don't know how many clocks correspond to a sleep of 1 microsecond.
            I could try through measurements to understand how many clocks correspond to 1 microsecond, but this is not so reliable. Different GPU can have different velocity (maybe one is running higher clock speed).

            clock_t start_clock = clock();
            clock_t clock_offset = 0;
            while (clock_offset < clock_count)
            {
                clock_offset = clock() - start_clock;
            }
            d_o[0] = clock_offset;

            */

        } else {

#if DEBUG_CUDA
            printf(
                            "MPI %d (out of %d). GPU: Started "
                            "to analize pattern n° %d.\n",
                            myRank, numberProcesses,
                            i);
#endif

            int sizeActualPattern = sizePatterns[i];

            int *column;
            column = (int *) malloc((sizeActualPattern + 1) * sizeof(int));
            if (column == NULL) {
                /*fprintf(
                        stderr,
                        "Error: unable to allocate memory for column (%ldB)\n",
                        (size_pattern + 1) * sizeof(int));
                // return 1;*/
            }

            // If I am not the last rank I should take in consideration
            // extra characters from the next piece: in this way I don't
            // miss words which are placed between two pieces. If am the
            // last rank I don't take extra characters as the other ranks
            // since the file is finished.
            int indexFinishMyPieceWithExtra =
                    indexFinishMyPieceWithoutExtra;
            if (myRank != numberProcesses - 1) {
                indexFinishMyPieceWithExtra += sizeActualPattern - 1;
            }

            // Traverse the input data up to the end of the file
            n_bytes = indexFinishMyPieceWithExtra;

            // It's not possible to parallelize with OpenMP this for since
            // the cycles are interconnected.
            int r;
            for (r = indexStartMyPiece; r < n_bytes - approx_factor; r++) {

                int distance = 0;
                int size;
                size = sizeActualPattern;
                if (n_bytes - r < sizeActualPattern) {
                    size = n_bytes - r;
                }

                // I cannot call directly levenshtein function
                // distance = levenshtein(pattern[i], &buf[r], size, column);

                unsigned int x, y, lastdiag, olddiag;

#pragma unroll
                for (y = 1; y <= size; y++) {
                    column[y] = y;
                }
#pragma unroll
                for (x = 1; x <= size; x++) {
                    column[0] = x;
                    lastdiag = x - 1;
                    for (y = 1; y <= size; y++) {
                        olddiag = column[y];
                        column[y] = MIN3(
                                column[y] + 1, column[y - 1] + 1,
                                lastdiag + (pattern[i][y - 1] == (&buf[r])[x - 1] ? 0 : 1));
                        lastdiag = olddiag;
                    }
                }

                distance = column[size];

                if (distance <= approx_factor) {
                    numbersOfMatch[i] += 1;

                }
            }

            free(column);
        }
    }

}


extern "C" int initializeGPU(char *buf, int n_bytes, char **pattern, int nb_patterns, int lastPatternAnalyzedByGPU,
                             int *sizePatterns, int indexFinishMyPieceWithoutExtra, int myRank, int numberProcesses,
                             int indexStartMyPiece, int approx_factor) {

#if DEBUG_CUDA
    printf("CUDA_DEBUG. Starting allocating data structures and memory transfers...\n");
#endif

    // Allocate space for the buffer and copy data
    char *d_buf;
    hipMalloc(&d_buf, n_bytes);
    hipMemcpy(d_buf, buf, n_bytes, hipMemcpyHostToDevice);

    // I need to know the size of patterns to copy the data. So I copy an array containing all the sizes of the patterns
    int *d_sizePatterns;
    hipMalloc(&d_sizePatterns, nb_patterns * sizeof(int));
    hipMemcpy(d_sizePatterns, sizePatterns, nb_patterns, hipMemcpyHostToDevice);

    // Allocate array where to save the number of matches
    hipMalloc(&d_numbersOfMatch, nb_patterns * sizeof(int));

    // Allocate array of patterns: that is an array of arrays
    char *d_pattern;
    hipMalloc(&d_pattern, nb_patterns * sizeof(char *));

    // Allocate space for each pattern and copy it
    for (int i = 0; i < nb_patterns; i++) {
        char *patternInUse = &d_pattern[i];
        hipMalloc(&patternInUse, sizePatterns[i] * sizeof(char));
        hipMemcpy(&d_pattern[i], pattern[i], sizePatterns[i], hipMemcpyHostToDevice);
    }

    int sizeGrid = 256;
    int sizeBlocks = 1;

#if DEBUG_CUDA
    printf("CUDA_DEBUG. Going to call the kernel code\n");
#endif

    searchPattern<<<sizeGrid, sizeBlocks>>>(d_buf, n_bytes, &d_pattern, nb_patterns, lastPatternAnalyzedByGPU,
                                            d_sizePatterns, d_numbersOfMatch, indexFinishMyPieceWithoutExtra, myRank,
                                            numberProcesses, indexStartMyPiece, approx_factor);

#if DEBUG_CUDA
    printf("CUDA_DEBUG. Kernel code returned.\n");
#endif

#if DEBUG_CUDA
    printf("CUDA_DEBUG. Copied results of CUDA.\n");
#endif

    return 1;

}

extern "C" int *
getGPUResult(int nb_patterns) {

    // Allocate local structure where to save the number of matches
    int *numbersOfMatch = (int *) malloc(nb_patterns * sizeof(int));

    hipMemcpy(numbersOfMatch, d_numbersOfMatch, nb_patterns * sizeof(int),
               hipMemcpyDeviceToHost);

    return numbersOfMatch;
}

// CUDA-C includes
#include <hip/hip_runtime.h>


#include <cstdio>

#define DEBUG_CUDA 1

extern "C" void getDeviceCount(int *deviceCountPtr) {
    hipError_t error_id = hipGetDeviceCount(deviceCountPtr);

    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
               static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
    return;
}

extern "C" void setDevice(int rank, int deviceCount) {
    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
    } else {
#ifdef DEBUG_CUDA
        printf(
            "Rank %d detected %d CUDA Capable device(s) - performing "
            "cudaSetDevice(0)\n",
            rank, deviceCount);
#endif
        hipSetDevice(0);
    }
    return;
}
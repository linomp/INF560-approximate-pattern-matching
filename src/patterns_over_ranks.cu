
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

// CUDA-C includes
#include <hip/hip_runtime.h>


#include <cstdio>

#define DEBUG_CUDA 1

#define MIN3(a, b, c) \
    ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

__global__ void ComputeMatches(char *buf, char *pattern, int *local_matches,
                               int n_bytes, int pattern_length,
                               int approx_factor) {
    // Source of tip about using pragma unroll:
    // https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#branch-predication

    // For loop inside kernel source:
    // https://www.diehlpk.de/blog/cuda-7-forall/

    int distance = 0;
    int j;

    int *column = (int *)malloc((pattern_length + 1) * sizeof(int));

    for (j = blockDim.x * blockIdx.x + threadIdx.x; j < n_bytes - approx_factor;
         j += gridDim.x * blockDim.x) {
        int size;

        size = pattern_length;
        if (n_bytes - j < pattern_length) {
            size = n_bytes - j;
        }

        // Levenshtein
        unsigned int x, y, lastdiag, olddiag;

#pragma unroll
        for (y = 1; y <= size; y++) {
            column[y] = y;
        }
#pragma unroll
        for (x = 1; x <= size; x++) {
            column[0] = x;
            lastdiag = x - 1;
            for (y = 1; y <= size; y++) {
                olddiag = column[y];
                column[y] = MIN3(
                    column[y] + 1, column[y - 1] + 1,
                    lastdiag + (pattern[y - 1] == (&buf[j])[x - 1] ? 0 : 1));
                lastdiag = olddiag;
            }
        }

        distance = column[size];

        if (distance <= approx_factor) {
            (*local_matches)++;
        }
    }

    free(column);
}

extern "C" int *invoke_kernel(char *buf, int n_bytes, char *my_pattern,
                              int pattern_length, int approx_factor,
                              int *local_matches) {
    // Allocate arrays in device memory
    char *d_buf;
    hipMalloc(&d_buf, n_bytes);
    char *d_pattern;
    hipMalloc(&d_pattern, pattern_length);
    int *d_local_matches;
    hipMalloc(&d_local_matches, 1 * sizeof(int));

#if DEBUG_CUDA
    printf("DEBUG_CUDA: Starting memory transfers...\n");
#endif

    // Copy buffer & pattern from host memory to device memory
    hipMemcpy(d_buf, buf, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_pattern, my_pattern, pattern_length, hipMemcpyHostToDevice);
    hipMemcpy(d_local_matches, local_matches, 1 * sizeof(int),
               hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n_bytes + threadsPerBlock - 1) / threadsPerBlock;

    ComputeMatches<<<blocksPerGrid, threadsPerBlock>>>(
        d_buf, d_pattern, d_local_matches, n_bytes, pattern_length,
        approx_factor);

#if DEBUG_CUDA
    printf("DEBUG_CUDA: Kernel invoked - &d_local_matches=%ld\n",
           d_local_matches);
#endif

    // Free device memory
    hipFree(d_buf);
    hipFree(d_pattern);

    return d_local_matches;
}

extern "C" void write_kernel_result(int *local_matches, int *d_local_matches) {
#if DEBUG_CUDA
    printf(
        "DEBUG_CUDA: getting result from device address &d_local_matches=%ld\n",
        d_local_matches);
#endif

    // Copy result from device memory to host memory
    hipMemcpy(local_matches, d_local_matches, 1 * sizeof(int),
               hipMemcpyDeviceToHost);

#if DEBUG_CUDA
    printf("DEBUG_CUDA: Matches found = %d\n", *local_matches);
#endif

    // Free remaining device pointer
    hipFree(d_local_matches);

    return;
}

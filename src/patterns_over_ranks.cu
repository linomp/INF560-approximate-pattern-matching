
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

// CUDA-C includes
#include <hip/hip_runtime.h>


#include <cstdio>

#define DEBUG_CUDA 1
#define TESTPERFORMANCE_NO_LEVENSHTEIN 1

#define MIN3(a, b, c) \
    ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

/* Function computing the final string to print */
__global__ void ComputeMatches(char *buf, char *pattern, int *local_matches,
                               int n_bytes, int pattern_length,
                               int approx_factor) {
    int distance = 0;

    int j = blockDim.x * blockIdx.x + threadIdx.x;

#if TESTPERFORMANCE_NO_LEVENSHTEIN
    // unsigned int ns = 8;
    //__nanosleep(ns);
    return;
#else
    int *column = (int *)malloc((pattern_length + 1) * sizeof(int));

    for (j = blockDim.x * blockIdx.x + threadIdx.x;
         j < (n_bytes / blockDim.x) - approx_factor; j++) {
        int size;

        size = pattern_length;
        if (n_bytes - j < pattern_length) {
            size = n_bytes - j;
        }

        // Levenshtein
        unsigned int x, y, lastdiag, olddiag;

        for (y = 1; y <= size; y++) {
            column[y] = y;
        }
        for (x = 1; x <= size; x++) {
            column[0] = x;
            lastdiag = x - 1;
            for (y = 1; y <= size; y++) {
                olddiag = column[y];
                column[y] = MIN3(
                    column[y] + 1, column[y - 1] + 1,
                    lastdiag + (pattern[y - 1] == (&buf[j])[x - 1] ? 0 : 1));
                lastdiag = olddiag;
            }
        }

        distance = column[size];

        if (distance <= approx_factor) {
            (*local_matches)++;
        }
    }

    free(column);
#endif
}

extern "C" int search_pattern_kernel(char *buf, int n_bytes, char *my_pattern,
                                     int pattern_length, int approx_factor,
                                     int *local_matches) {
    // Allocate arrays in device memory
    char *d_buf;
    hipMalloc(&d_buf, n_bytes);
    char *d_pattern;
    hipMalloc(&d_pattern, pattern_length);
    int *d_local_matches;
    hipMalloc(&d_local_matches, 1 * sizeof(int));

#if DEBUG_CUDA
    printf("DEBUG_CUDA: Starting memory transfers...\n");
#endif

    // Copy buffer & pattern from host memory to device memory
    hipMemcpy(d_buf, buf, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_pattern, my_pattern, pattern_length, hipMemcpyHostToDevice);
    hipMemcpy(d_local_matches, local_matches, 1 * sizeof(int),
               hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n_bytes + threadsPerBlock - 1) / threadsPerBlock;

    ComputeMatches<<<blocksPerGrid, threadsPerBlock>>>(
        d_buf, d_pattern, d_local_matches, n_bytes, pattern_length,
        approx_factor);

    // Copy result from device memory to host memory
    hipMemcpy(local_matches, d_local_matches, 1 * sizeof(int),
               hipMemcpyDeviceToHost);

#if DEBUG_CUDA
    printf("DEBUG_CUDA: Matches found in the first %d bytes = %d\n", n_bytes,
           *local_matches);
#endif

    // Free device memory
    hipFree(d_buf);
    hipFree(d_pattern);
    hipFree(local_matches);

    return 0;
}

/*
#pragma omp parallel default(none)                                         \
    firstprivate(rank, n_bytes, approx_factor, pattern_length, my_pattern, \
                 cuda_device_exists) shared(buf, local_matches)
{
    rank = rank;

    // Overall idea: if there is a cuda device, omp threads take on
    // just half of the workload + "ghost cells"
    n_bytes =
        cuda_device_exists ? ((n_bytes / 2) + (pattern_length - 1)) :
n_bytes;

    approx_factor = approx_factor;
    pattern_length = pattern_length;
    my_pattern = my_pattern;

    int j;

    int chunk_size = (2 * pattern_length) - 1;  // offset for ghost cells

    int *column = (int *)malloc((chunk_size + 1) * sizeof(int));

#if APM_DEBUG
    printf("thread: %d - chunk_size: %d\n", omp_get_thread_num(),
chunk_size); #endif printf("Starting with local matches = %d\n",
local_matches);

#pragma omp for schedule(dynamic, chunk_size)
    for (j = 0; j < n_bytes - approx_factor; j++) {
#if APM_DEBUG_BYTES
        printf("(Rank %d - Thread %d) - processing byte %d\n", rank,
               omp_get_thread_num(), j);
#endif
        int distance = 0;
        int size;

        size = pattern_length;
        if (n_bytes - j < pattern_length) {
            size = n_bytes - j;
        }

        distance = levenshtein(my_pattern, &buf[j], size, column);

        if (distance <= approx_factor) {
            local_matches++;
        }
    }
    free(column);
}*/